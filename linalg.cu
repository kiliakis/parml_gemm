#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "linalg.h"
#include <stdio.h>

#ifdef CUDA
extern hipblasHandle_t cublas_handle;
#endif

// #define TILE_DIM 32
// #define TILE_N 16
// #define TILE_TB_HEIGHT 8
// #define TILE_M (TILE_N*TILE_TB_HEIGHT)

// int BLOCK_SIZE = 32;

/*
 *  Naive matrix multiply kernels.
 */
__global__ void dgemm_naive(const double *A, const double *B,
                            double *C,
                            const int M, const int N, const int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        double sum = 0.;
        for (int k = 0; k < K; k++)
            sum += A[row * K + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

__global__ void dgemm_ta_naive(const double *A, const double *B,
                               double *C,
                               const int M, const int N, const int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        double sum = 0;
        for (int k = 0; k < K; k++)
            sum += A[k * M + row] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

__global__ void dgemm_tb_naive(const double *A, const double *B, const double *C,
                               double *D,
                               const int M, const int N, const int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        double sum = 0;
        for (int k = 0; k < K; k++)
            sum += A[row * K + k] * B[col * K + k];
        D[row * N + col] = sum + C[row * N + col];
    }
}

/*
 *  Optimized matrix multiply kernels using shared memory.
 */
// A: M x K, B: K x N, C: M x N
template <int BLOCK_SIZE> __global__ void dgemm_optimized(
    const double *A, const double *B,
    double *C,
    const int M, const int N, const int K)
{
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    double Csub = 0;

    // Declaration of the shared memory array As, Bs used to
    // store the sub-matrix of A, B
    __shared__ double shmem[2 * BLOCK_SIZE * BLOCK_SIZE];
    double *As = shmem;
    double *Bs = &(shmem[BLOCK_SIZE*BLOCK_SIZE]); 

    for (int k = 0; k < (BLOCK_SIZE + K -1); k+=BLOCK_SIZE) {

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        if (k + threadIdx.x < K && row < M)
            // As[threadIdx.y][threadIdx.x] = A[row*K + k*BLOCK_SIZE + threadIdx.x];
            As[threadIdx.y* BLOCK_SIZE + threadIdx.x] = A[row*K + k + threadIdx.x];
        else
            // As[threadIdx.y][threadIdx.x] = 0.0;
            As[threadIdx.y* BLOCK_SIZE + threadIdx.x] = 0.0;

        if (k + threadIdx.y < K && col < N)
            // Bs[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE + threadIdx.y)*N + col];
            Bs[threadIdx.y * BLOCK_SIZE + threadIdx.x] = B[(k + threadIdx.y)*N + col];
        else
            // Bs[threadIdx.y][threadIdx.x] = 0.0;
            Bs[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0;

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        #pragma unroll BLOCK_SIZE
        for (int n = 0; n < BLOCK_SIZE; ++n) {
            Csub += As[threadIdx.y*BLOCK_SIZE + n] * Bs[n*BLOCK_SIZE+threadIdx.x];
            // Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    if (row < M && col < N)
        C[row*N + col] = Csub;
}

// A: K x M, B: K x N, C: M x N
template <int BLOCK_SIZE> __global__ void dgemm_ta_optimized(const double *A, const double *B,
                                   double *C,
                                   const int M, const int N, const int K) {

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    double Csub = 0;

    // Declaration of the shared memory array As, Bs used to
    // store the sub-matrix of A, B
    __shared__ double shmem[2 * BLOCK_SIZE * BLOCK_SIZE];
    double *As = shmem;
    double *Bs = &(shmem[BLOCK_SIZE*BLOCK_SIZE]); 

    for (int k = 0; k < (BLOCK_SIZE + K -1); k+=BLOCK_SIZE) {

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        // if (k + threadIdx.x < K && row < M)
        if (k + threadIdx.x < K && row < M)
            // As[threadIdx.y* BLOCK_SIZE + threadIdx.x] = A[row*K + k + threadIdx.x];
            As[threadIdx.y* BLOCK_SIZE + threadIdx.x] = A[row + (k + threadIdx.x)*M];
        else
            // As[threadIdx.y][threadIdx.x] = 0.0;
            As[threadIdx.y* BLOCK_SIZE + threadIdx.x] = 0.0;

        if (k + threadIdx.y < K && col < N)
            // Bs[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE + threadIdx.y)*N + col];
            Bs[threadIdx.y * BLOCK_SIZE + threadIdx.x] = B[(k + threadIdx.y)*N + col];
        else
            // Bs[threadIdx.y][threadIdx.x] = 0.0;
            Bs[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0;

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        #pragma unroll BLOCK_SIZE
        for (int n = 0; n < BLOCK_SIZE; ++n) {
            Csub += As[threadIdx.y*BLOCK_SIZE + n] * Bs[n*BLOCK_SIZE+threadIdx.x];
            // Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    if (row < M && col < N)
        C[row*N + col] = Csub;
}

__global__ void dgemm_tb_optimized(const double *A, const double *B, const double *C,
                                   double *D,
                                   const int M, const int N, const int K) {
    /*
     * FILLME: fill the code.
     */
}

// Computes C = A*B, where A is a M by K matrix, B is a K by N matrix, C is a M by N matrix.
// Matrices are stored in row-major order.
void dgemm_gpu(const double *A, const double *B, double *C, const int M, const int N, const int K) {
#ifndef CUBLAS
    const int BLOCK_SIZE = 32;
#if defined(_GPU_GEMM_NAIVE)
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dgemm_naive <<< grid, block>>>(A, B, C, M, N, K);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#elif defined(_GPU_GEMM_OPT)
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + block.x - 1) / block.x,
              (M + block.y - 1) / block.y);
    size_t shmem_size = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    dgemm_optimized<BLOCK_SIZE> <<< grid, block, shmem_size>>>(A, B, C, M, N, K);

    // dim3 block(TILE_N, TILE_TB_HEIGHT);
    // dim3 grid(M/TILE_M, N/TILE_N);
    // size_t shmem_size = TILE_TB_HEIGHT * TILE_N * sizeof(double);
    // dgemm_optimized <<< grid, block, shmem_size>>>(B, A, C, N, M, K);

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#endif
#else
// Matrices are stored in row-major order, but cuBLAS assumes column-major
// order. We want to compute:
//         A * B = (A^T)^T * (B^T)^T = A'^T * B'^T = (B' * A')^T
    /*
     *  FILLME: Use hipblasDgemm()
     */
    hipblasStatus_t stat;
    double alpha = 1;
    double beta = 0;
    stat = hipblasDgemm(cublas_handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       N, M, K,
                       &alpha,
                       B, N,
                       A, K,
                       &beta,
                       C, N);
    checkCublasErrors(stat);
#endif
}

// Computes C = A'*B, where A is a K by M matrix, B is a K by N matrix, C is a M by N matrix.
// Matrices are stored in row-major order.
void dgemm_ta_gpu(const double *A, const double *B, double *C, const int M, const int N, const int K) {
#ifndef CUBLAS
    const int BLOCK_SIZE = 16;
#if defined(_GPU_GEMM_NAIVE)
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dgemm_ta_naive <<< grid, block>>>(A, B, C, M, N, K);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#elif defined(_GPU_GEMM_OPT)

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + block.x - 1) / block.x,
              (M + block.y - 1) / block.y);
    size_t shmem_size = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    dgemm_ta_optimized<BLOCK_SIZE> <<< grid, block, shmem_size>>>(A, B, C, M, N, K);

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#endif
#else
// Matrices are stored in row-major order, but cuBLAS assumes column-major
// order. We want to compute:
//         A^T * B = A^T * (B^T)^T = A' * B'^T = (B'*A'^T)^T
    /*
     *  FILLME: Use hipblasDgemm()
     */
    hipblasStatus_t stat;
    double alpha = 1;
    double beta = 0;
    stat = hipblasDgemm(cublas_handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_T,
                       N, M, K,
                       &alpha,
                       B, N,
                       A, M, // A, M
                       &beta,
                       C, N);
    checkCublasErrors(stat);

#endif
}

// Computes D = A*B'+C, where A is a M by K matrix, B is a N by K matrix, C and D are M by N matrices.
// Matrices are stored in row-major order.
void dgemm_tb_gpu(const double *A, const double *B, const double *C, double *D, const int M, const int N, const int K) {
#ifndef CUBLAS
    const int BLOCK_SIZE = 16;
#if defined(_GPU_GEMM_NAIVE)
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dgemm_tb_naive <<< grid, block>>>(A, B, C, D, M, N, K);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#elif defined(_GPU_GEMM_OPT)
    /*
     *  FILLME: Set up the blocks, grid and the shared memory size.
     */
    dim3 block(1, 1);
    dim3 grid(1, 1);
    size_t shmem_size = 0;
    dgemm_tb_optimized <<< grid, block, shmem_size>>>(A, B, C, D, M, N, K);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
#endif
#else
// D = A * B^T
// Matrices are stored in row-major order, but cuBLAS assumes column-major
// order. We want to compute:
//         C = A * B^T = (A^T)^T * B^T  = A'^T * B' = (B'^T * A')^T
    /*
     *  FILLME: Use hipblasDgemm()
     */
    hipblasStatus_t stat;
    double alpha = 1;
    double beta = 0;
    stat = hipblasDgemm(cublas_handle,
                       HIPBLAS_OP_T, HIPBLAS_OP_N,
                       N, M, K,
                       &alpha,
                       B, K, // B, K
                       A, K,
                       &beta,
                       D, N);
    checkCublasErrors(stat);

// D = C + D
    /*
     *  FILLME: Use hipblasDgeam()
     */
    // C in row-major is M x N, so in col-major it is N x M
    alpha = 1;
    beta = 1;
    stat = hipblasDgeam(cublas_handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       M, N,
                       &alpha,
                       C, M,
                       &beta,
                       D, M,
                       D, M);
    checkCublasErrors(stat);

#endif
}


/*
 *  Optimized matrix multiply kernels using shared memory.
 */
// __global__ void dgemm_optimized(const double *A, const double *B,
// double *C,
// const int M, const int N, const int K) {
/*
 * FILLME: fill the code.
 */

/*
// from https://stackoverflow.com/questions/18815489/cuda-tiled-matrix-matrix-multiplication-with-shared-memory-and-matrix-size-whic/18856054
double CValue = 0;
const int ARows = M;
const int ACols = K;
const int BRows = K;
const int BCols = N;
const int CRows = M;
const int CCols = N;

int Row = blockIdx.y*TILE_DIM + threadIdx.y;
int Col = blockIdx.x*TILE_DIM + threadIdx.x;

__shared__ double As[TILE_DIM][TILE_DIM];
__shared__ double Bs[TILE_DIM][TILE_DIM];

for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

     if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
         As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
     else
         As[threadIdx.y][threadIdx.x] = 0.0;

     if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
         Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
     else
         Bs[threadIdx.y][threadIdx.x] = 0.0;

     __syncthreads();

     for (int n = 0; n < TILE_DIM; ++n)
         CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

     __syncthreads();
}

if (Row < CRows && Col < CCols)
    C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
       (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
*/
// from parboil sgemm
/*
// Partial results
double c[TILE_N];
for (int i=0; i < TILE_N; i++) c[i] = 0.0;
int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
int m = blockIdx.x * TILE_M + mid;
int n = blockIdx.y * TILE_N + threadIdx.x;
__shared__ double b_s[TILE_TB_HEIGHT][TILE_N];
for (int i = 0; i < K; i+=TILE_TB_HEIGHT) {
    double a;
    b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*N];
    __syncthreads();
    for (int j = 0; j < TILE_TB_HEIGHT; j++) {
        a = A[m + (i+j)*K];
        for (int kk = 0; kk < TILE_N; kk++)
            c[kk] += a * b_s[j][kk];
    }
    __syncthreads();
}
int t = N*blockIdx.y * TILE_N + m;
for (int i = 0; i < TILE_N; i++) {
    // C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
    C[t+i*N] = c[i];
}
*/

// }
