#include "cuda_utils.h"
#include <stdio.h>

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line) {
  if (result) {
    printf("CUDA error at %s: %d code = %d (%s) %s", file, line,
           static_cast<unsigned int>(result), hipGetErrorName(result), func);
    exit(EXIT_FAILURE);
  }
}

// cuBLAS API errors
static const char *_cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";

  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";

  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";

  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";

  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";

  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";

  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";

  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";

  case HIPBLAS_STATUS_NOT_SUPPORTED:
    return "HIPBLAS_STATUS_NOT_SUPPORTED";

  case HIPBLAS_STATUS_UNKNOWN:
    return "HIPBLAS_STATUS_UNKNOWN";
  }

  return "<unknown>";
}

void check_cublas(hipblasStatus_t result, char const *const func,
                  const char *const file, int const line) {
  if (result) {
    printf("cuBLAS error at %s: %d code=%d (%s) %s\n", file, line,
           (unsigned int)result, _cublasGetErrorString(result), func);
    exit(EXIT_FAILURE);
  }
}
